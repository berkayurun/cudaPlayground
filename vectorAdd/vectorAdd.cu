
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(float *x, float *y){
	int index = threadIdx.x;

	x[index] += y[index];
}

void printVectors(int n, float *x, float *y){
	std::cout << "Printing x - y: " << std::endl;
	for(int i = 0; i < n; i++){
		std::cout << x[i] << " - " << y[i] << std::endl;
	}
}

int main(){
	int n = 10;
	float *x;
	float *y;

	hipMallocManaged(&x, n * sizeof(float));
	hipMallocManaged(&y, n * sizeof(float));

	for(int i = 0; i < n; i++){
		x[i] = 1.0f;
		y[i] = 3.5f;
	}

	printVectors(n, x, y);

	add<<<1, n>>>(x, y);

	hipDeviceSynchronize();

	printVectors(n, x, y);
}

